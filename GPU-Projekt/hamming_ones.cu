#include "hip/hip_runtime.h"
﻿#include "hamming_ones.h"

#if defined (__INTELLISENSE__) | defined (__RESHARPER__)
template<class T1, class T2>
__device__ void atomicAdd(T1 x, T2 y);
#endif

int main()
{
    unsigned int* sequences = (unsigned int*)malloc(NUMBER * LENGTH * sizeof(unsigned int));
    GenerateSequences(sequences);
    unsigned int* result = (unsigned int*)malloc(sizeof(unsigned int));

    if (FORCE_ONE_PAIR) {
        CreateOnePair(sequences);
    }
    
    auto start = std::chrono::high_resolution_clock::now();
    if (CPU) {
        GetHammingOnes(sequences, result);
    }
    else {
        unsigned int* d_sequences;
        unsigned int* d_result;
        hipMalloc(&d_sequences, NUMBER * LENGTH * sizeof(unsigned int));
        hipMalloc(&d_result, sizeof(unsigned int));
        hipMemcpy(d_sequences, sequences, NUMBER * LENGTH * sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(d_result, result, sizeof(unsigned int), hipMemcpyHostToDevice);
        unsigned int blocks = (NUMBER + THREAD_COUNT - 1) / THREAD_COUNT;

        if (HASH) {
            int* keys = (int*)malloc(HASH_MAP_SIZE * sizeof(int));
            unsigned int* values = (unsigned int*)malloc(HASH_MAP_SIZE * sizeof(unsigned int) * LENGTH);
            int* d_keys;
            unsigned int* d_values;

            SetUpHashTable(keys, values, sequences);
            hipMalloc(&d_keys, HASH_MAP_SIZE * sizeof(int));
            hipMalloc(&d_values, HASH_MAP_SIZE * sizeof(unsigned int) * LENGTH);
            hipMemcpy(d_keys, keys, HASH_MAP_SIZE * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_values, values, HASH_MAP_SIZE * sizeof(unsigned int) * LENGTH, hipMemcpyHostToDevice);

            GetHammingOnesGPUHash << <blocks * LENGTH, THREAD_COUNT >> > (d_sequences, d_result, d_keys, d_values);
            hipFree(d_keys);
            hipFree(d_values);
            free(keys);
            free(values);
        }
        else {
            GetHammingOnesGPU << <blocks, THREAD_COUNT >> > (d_sequences, d_result);
        }

        hipMemcpy(result, d_result, sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipFree(d_sequences);
        hipFree(d_result);
    }
    auto finish = std::chrono::high_resolution_clock::now();
    float seconds = (float)(std::chrono::duration_cast<std::chrono::microseconds>(finish - start).count()) / 1000000.0;
        
    PrintResults(result[0], seconds);

    free(sequences);
    free(result);
    return 0;
}

void CreateOnePair(unsigned int* sequences) {
    for (int i = 0; i < LENGTH; i++) {
        sequences[69 * LENGTH + i] = sequences[420 * LENGTH + i];
    }
    sequences[69 * LENGTH] ^= 1UL << 4;
}

void GetHammingOnes(unsigned int* sequences, unsigned int* result) {
    result[0] = 0;
    for (int i = 0; i < NUMBER; i++) {
        for (int j = i + 1; j < NUMBER; j++) {
            if (CheckIfHammingOnes(&sequences[i * LENGTH], &sequences[j * LENGTH])) {
                PrintPair(&sequences[i * LENGTH], i, &sequences[j * LENGTH], j);
                result[0]++;
            }
        }
    }
}

void GenerateRandomBits(unsigned int* sequence) {
    unsigned int start = ((unsigned int)rand()) << 17;
    unsigned int middle = ((unsigned int)rand()) << 2;
    unsigned int end = rand() % 3;
    *sequence = start + middle + end;
}

void GenerateSequences(unsigned int* sequences) {
    srand(SEED);

    for (int i = 0; i < NUMBER * LENGTH; i++) {
        GenerateRandomBits(&sequences[i]);
    }
}

__host__ __device__ unsigned int CountSetBits(unsigned int n)
{
    unsigned int count = 0;
    while (n) {
        count += n & 1;
        n >>= 1;
    }
    return count;
}

__host__ __device__ bool CheckIfHammingOnes(unsigned int* s1, unsigned int* s2) {
    int counter = 0;
    for (int i = 0; i < LENGTH; i++) {
        unsigned int xor = s1[i] ^ s2[i];
        counter += CountSetBits(xor);

        if (counter > 1) return false;
    }

    if (counter == 1) return true;
    return false;
}

__host__ __device__ void PrintBits(unsigned int num) {
    int size = sizeof(unsigned int);
    unsigned int maxPow = 1 << (size * 8 - 1);
    for (int i = 0; i < size; ++i) {
        for (; i < size * 8; ++i) {
            // print last bit and shift left.
            printf("%u ", num & maxPow ? 1 : 0);
            num = num << 1;
        }
    }
}

__host__ __device__ void PrintSequence(unsigned int* sequence) {
    for (int i = 0; i < LENGTH; i++) {
        PrintBits(sequence[i]);
    }
}

__host__ __device__ void PrintPair(unsigned int* s1, int i, unsigned int* s2, int j) {
    printf("%7d: ", i);
    PrintSequence(s1);
    printf("\n");
    printf("%7d: ", j);
    PrintSequence(s2);
    printf("\n==========\n");
}

unsigned int CountResultNumber(unsigned int* result) {
    unsigned int sum = 0;
    for (int i = 0; i < NUMBER; i++) {
        sum += result[i];
    }

    return sum;
}

void PrintResults(unsigned int result_number, float seconds) {
    printf("\n");
    printf("The program found %d results. It took %f seconds", result_number, seconds);
}

__global__ void GetHammingOnesGPU(unsigned int* sequences, unsigned int* result) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= NUMBER) return;
    unsigned int* main = &sequences[id * LENGTH];
    for (int i = id + 1; i < NUMBER; i++) {
        if (CheckIfHammingOnes(main, &sequences[i * LENGTH])) {
            printf("%d - %d\n", id, i);
            atomicAdd(&result[0], 1);
        }
    }
}

__device__ unsigned int CountSetBitsGPU(unsigned int n)
{
    unsigned int count = 0;
    while (n) {
        count += n & 1;
        n >>= 1;
    }
    return count;
}

__device__ bool CheckIfHammingZerosGPU(unsigned int* s1, unsigned int* s2) {
    for (int i = 0; i < LENGTH; i++) {
        unsigned int xor = s1[i] ^ s2[i];
        if (CountSetBitsGPU(xor) > 0) return false;
    }

    return true;
}

__global__ void GetHammingOnesGPUHash(unsigned int* sequences, unsigned int* result, int* keys, unsigned int* values) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= NUMBER * LENGTH) return;

    int i = id % LENGTH;
    int index = id - i;
    if(id == 0) result[0] = 0;
    unsigned int seq[LENGTH];

    for (int j = 0; j < LENGTH; j++) {
        seq[j] = sequences[index + j];
    }

    for (int j = 0; j < 32; j++) {
        seq[i] ^= 1UL << j;
        int key = GetKey(keys, values, seq);
                
        if (key != -1) {
            if (key < id / LENGTH) {
                printf("%d - %d\n", id / LENGTH, key);
                atomicAdd(&result[0], 1);
            }
        }
        seq[i] ^= 1UL << j;
    }
}

void Add(int* keys, unsigned int* values, int key, unsigned int* seq) {
    unsigned int i = HashSequence(seq);
    while (keys[i] != 0) {
        i = (i + 1) % HASH_MAP_SIZE;
    }

    keys[i] = key;
    for (int j = 0; j < LENGTH; j++) {
        values[i * LENGTH + j] = seq[j];
    }
}

__host__ __device__ unsigned int HashSequence(unsigned int* seq) {
    unsigned int result = 0;

    for (int i = 0; i < LENGTH; i++) {
        result = result ^ Hash(seq[i]);
    }

    return result % HASH_MAP_SIZE;
}

__host__ __device__ unsigned int Hash(unsigned int x) {
    x = ((x >> 16) ^ x) * 0x45d9f3b;
    x = ((x >> 16) ^ x) * 0x45d9f3b;
    x = (x >> 16) ^ x;

    return x;
}

void SetUpHashTable(int* keys, unsigned int* values, unsigned int* sequences) {
    SetUpKeys(keys);
    for (int i = 0; i < NUMBER; i++) {
        Add(keys, values, i, &sequences[i * LENGTH]);
    }
}

void SetUpKeys(int* keys) {
    for (int i = 0; i < HASH_MAP_SIZE; i++) {
        keys[i] = 0;
    }
}

__device__ int GetKey(int* keys, unsigned int* values, unsigned int* sequence) {
    int i = HashSequence(sequence);
    
    while (keys[i] != 0) {
        if (CheckIfHammingZerosGPU(&values[i * LENGTH], sequence)) return keys[i];

        i = (i + 1) % HASH_MAP_SIZE;
    }

    return -1;
}